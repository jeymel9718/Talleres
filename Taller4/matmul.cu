#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

int *a, *b;  // host data
int *c, *c2;  // results

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0);
 }
}

//GPU kernel
__global__
void matMul(int *A,int *B,int *C,int N){
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    float tmpSum = 0;
    if (ROW < N && COL < N) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
     C[ROW*N + COL] = tmpSum;
    }
}

//CPU function
void matMul_h(int *A1,int *B1, int *C1, int N){
    float tmpSum=0;
    for(int i = 0; i < N; i++){
      for(int j = 0; j < N;++j){
       for (int k = 0; k < N;++k){
         tmpSum+=A1[i*N + k] + B1[k*N + j];
       }
       C1[i*N + j] = tmpSum;
       tmpSum = 0;
      }
    }
}

int main(int argc,char **argv)
{
    printf("Begin \n");
    //N*N matrix
    int n=4*4;
    //Number of blocks
    int nBytes = n*sizeof(int);
    //Block size and number
    int block_size, block_no;

    //memory allocation
    a = (int *) malloc(nBytes);
    b = (int *) malloc(nBytes);
    c = (int *) malloc(nBytes);
    c2 = (int *) malloc(nBytes);

    int *a_d,*b_d,*c_d;
    block_size = 4; //threads per block
    block_no = 1;

    //Work definition
    dim3 dimBlock(block_size, block_size, 1);
    dim3 dimGrid(block_no, block_no, 1);

    // Data filling
    for(int i=0;i<n;i++)
    a[i]=i,b[i]=i;


    printf("Allocating device memory on host..\n");
   //GPU memory allocation
    hipMalloc((void **) &a_d, n*sizeof(int));
    hipMalloc((void **) &b_d, n*sizeof(int));
    hipMalloc((void **) &c_d, n*sizeof(int));

    printf("Copying to device..\n");
    hipMemcpy(a_d, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, n*sizeof(int), hipMemcpyHostToDevice);

    clock_t start_d=clock();
    printf("Doing GPU Vector add\n");
    matMul<<<block_no,block_size>>>(a_d, b_d, c_d, n);
    cudaCheckError();

    //Wait for kernel call to finish
    hipDeviceSynchronize();

    clock_t end_d = clock();


    printf("Doing CPU Vector add\n");
    clock_t start_h = clock();
    matMul_h(a, b, c2, n);
    clock_t end_h = clock();

    //Time computing
    double time_d = (double)(end_d-start_d)/CLOCKS_PER_SEC;
    double time_h = (double)(end_h-start_h)/CLOCKS_PER_SEC;

    //Copying data back to host, this is a blocking call and will not start unt$
    hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost);
    printf("n = %d \t GPU time = %fs \t CPU time = %fs\n", n, time_d, time_h);

    //Free GPU memory
     hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
    return 0;
}




